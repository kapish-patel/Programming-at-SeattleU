/**
 * reduce_scan_1block.cu - using dissemination reduction for reducing and scanning a small array with CUDA
 * Kevin Lundeen, Seattle University, CPSC 5600 demo program
 * Notes:
 * - only works for one block (maximum block size for all of CUDA is 1024 threads per block)
 * - eliminated the remaining data races that were in reduce_scan_naive.cu
 * - algo requires power of 2 so we pad with zeros up to 1024 elements
 * - now a bit faster using block shared memory during loops (which also handily exposed the data races we had before)
 */


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int MAX_BLOCK_SIZE = 1024;

__global__ void allreduce(float *data) {
    __shared__ float local[MAX_BLOCK_SIZE]; // 10x faster at least than global memory via data[]
    int gindex = threadIdx.x;
    int index = gindex;
    local[index] = data[gindex];
    for (int stride = 1; stride < blockDim.x; stride *= 2) {

        __syncthreads();  // wait for my writing partner to put his value in local before reading it
        int source = (index - stride) % blockDim.x;
        float addend = local[source];

        __syncthreads();  // wait for my reading partner to pull her value from local before updating it
        local[index] += addend;
    }
    data[gindex] = local[index];
}

__global__ void scan(float *data) {
    __shared__ float local[MAX_BLOCK_SIZE];
    int gindex = threadIdx.x;
    int index = gindex;
    local[index] = data[gindex];
    for (int stride = 1; stride < blockDim.x; stride *= 2) {

        __syncthreads();  // cannot be inside the if-block 'cuz everyone has to call it!
        int addend = 0;
        if (stride <= index)
            addend = local[index - stride];

        __syncthreads();
        local[index] += addend;
    }
    data[gindex] = local[index];
}

void fillArray(float *data, int n, int sz) {
    for (int i = 0; i < n; i++)
        data[i] = (float)(i+1); // + (i+1)/1000.0;
    for (int i = n; i < sz; i++)
        data[i] = 0.0; // pad with 0.0's for addition
}

void printArray(float *data, int n, string title, int m=5) {
    cout << title << ":";
    for (int i = 0; i < m; i++)
        cout << " " << data[i];
    cout << " ...";
    for (int i = n - m; i < n; i++)
        cout << " " << data[i];
    cout << endl;
}

int main(void) {
    int n;
    float *data;
    int threads = MAX_BLOCK_SIZE;
    cout << "How many data elements? ";
    cin >> n;
    if (n > threads) {
        cerr << "Cannot do more than " << threads << " numbers with this simple algorithm!" << endl;
        return 1;
    }
    hipMallocManaged(&data, threads * sizeof(*data));
    fillArray(data, n, threads);
    printArray(data, n, "Before");
    allreduce<<<1, threads>>>(data);
    hipDeviceSynchronize();
    printArray(data, n, "Reduce");
    fillArray(data, n, threads);
    scan<<<1, threads>>>(data);
    hipDeviceSynchronize();
    printArray(data, n, "Scan");
    return 0;
}
