/**
 * @file hw6.cu - parallely sort x values and perform parallel scan of y values of a data in csv file.
 * @author Kapish Patel
 * @see "Seattle University, CPSC5600, Winter 2023"
 * @note i have implemented bitonic sort and paralle scan of values in a csv file i have used CUDA to complete this assignment.
    * Step 1: read csv with padding at end if not power of 2 in structure object array
    * Step 2: use bitonic sort to sort x values
    * Step 3: make parallel scan for y values
    * Step 4: Write a csv file from that structure object array
 * OUTPUT:
        1048576 Rows found in: x_y.csv
        Total number of Blocks required: 1024
        Data Sorted Properly.....
        1048576 Rows written in file: Result.csv
        * * *  Memory Free * * *
 * GPU:
        Type       Time(%)   Time        Calls   Avg        Min       Max         Name
 GPU activities:   50.86%  39.902ms       210  190.01us  148.25us  377.04us  bitonic(Data*, int, int)
                   35.21%  27.628ms         1  27.628ms  27.628ms  27.628ms  accum(Data*, float*, int, int)
                    8.36%  6.5619ms         2  3.2810ms  3.2533ms  3.3086ms  [CUDA memcpy DtoH]
                    4.61%  3.6173ms         1  3.6173ms  3.6173ms  3.6173ms  [CUDA memcpy HtoD]
                    0.95%  748.97us         1  748.97us  748.97us  748.97us  scan(Data*, float*, int, int)
      API calls:   55.27%  101.94ms         2  50.971ms  234.71us  101.71ms  cudaMalloc
                   37.32%  68.828ms       212  324.66us  150.88us  27.630ms  cudaDeviceSynchronize
                    6.03%  11.129ms         3  3.7097ms  3.6366ms  3.8291ms  cudaMemcpy
                    0.64%  1.1874ms         2  593.72us  516.46us  670.98us  cudaFree
*/

//Header files 

#include <hip/hip_runtime.h>
#include <sys/fcntl.h>  
#include <fcntl.h>
#include <sys/file.h>
#include <iostream>
#include <stdio.h>
#include <sstream>
#include <vector>
#include <fstream>
#include <iomanip>
#include <cfloat>
#include <cmath>

using namespace std;

//Global Variables
const int MAX_BLOCK_SIZE = 1024; // Constant all cuda blocks have 1024 threads
const string filename = "x_y.csv";
const string output_filename = "Result.csv";

//Structure Data to hold data from the csv
struct Data
{
    float x;
    float y;
    int row_num;
    float cumulative_y;
};

// Device swap function which swap values of structure 
__device__ void swap(Data *data, int a, int b) 
{
	Data temp = data[a];
	data[a] = data[b];
	data[b] = temp;
}

//Device Bitonic Function which sort the values in a structure
__global__ void bitonic(Data *values, int j, int k) 
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
  
    int ixj = i ^ j;

    if ((ixj)>i) 
    {
        if ((i&k)==0) 
        {
            //Sort ascending 
            if (values[i].x>values[ixj].x) 
                swap(values, i, ixj);
        }
        if ((i&k)!=0) 
        {
            //Sort descending 
            if (values[i].x<values[ixj].x)
                swap(values, i, ixj); 
        }
    }
}

//Device scan Function which perform a scan operation on y values
__global__ void scan(Data * data, float * sums, int n, int block_size) 
{
    int block_id = blockIdx.x; //block Id
    int thread_id = threadIdx.x; //thread id
    int idx = block_id * block_size + thread_id; //global index
    if (idx < n) 
        data[idx].cumulative_y = data[idx].y;
    else 
        data[idx].cumulative_y = 0.0;
    __syncthreads();

    //compute prefix sum of the current block 
    for (int stride = 1; stride < blockDim.x; stride <<= 1) 
    {
        __syncthreads();
        int prev_idx = idx - stride;
        if (prev_idx >= block_id * block_size) 
            data[idx].cumulative_y += data[prev_idx].cumulative_y;
    }
    if (thread_id == blockDim.x - 1) 
        sums[block_id] = data[idx].cumulative_y;
    __syncthreads();
}
  
  //accumulate prefix sum parallely
__global__ void accum(Data * data, float * sums, int n, int block_size) 
{
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;
    int idx = block_id * block_size + thread_id;

    // accumulate the prefix sums of previous blocks
    for (int i = 0; i < block_id; i++) 
    {
        if (block_id > 0) 
            data[idx].cumulative_y += sums[i];
    }
}


//Function declaration
int nearestPowerOfTwo(int n);
void skipLines(fstream& file, int numLines); 
int calcrows(string file);
Data* ReadCSV(int& number_of_rows);
void printdata(Data* data, int& number_of_rows);
void writeCSV(Data* data, int& number_of_rows, string filename);
void validate_Data(Data* data, int& number_of_rows);

// Host function or Main function()
int main()
{
    // code for mutex lock tells if someone is locked out...
    if (flock(open("/home/fac/lundeenk/hw6.lock", O_RDONLY), LOCK_EX|LOCK_NB) == -1) 
    {
        std::cout << "someone has hw6 locked--could it be you?" << std::endl;
        exit(1);
    }

    //Calculate number or rows and nearest power of 2
    int number_of_rows = calcrows(filename);
    int nearest_pow2 = nearestPowerOfTwo(number_of_rows);
    cout<<number_of_rows<<" Rows found in: "<<filename<<endl;
    
    //calculate size of array
    size_t array_Size_bytes = nearest_pow2*sizeof(Data);

    //Allocate memory to host
    Data* data = (Data*)malloc(array_Size_bytes);

    //allocate memory to Device
    Data* d_data;
    hipMalloc(&d_data , array_Size_bytes);

    //Read the data from CSVfile and store in data
    data = ReadCSV(number_of_rows);

    //Copy the data from host to device memory
    hipMemcpy(d_data, data, array_Size_bytes, hipMemcpyHostToDevice);

    //calculating number of threads and blocks required 
    int threads_per_Block = MAX_BLOCK_SIZE;
    int block_in_Grid = (number_of_rows + (MAX_BLOCK_SIZE - 1)) / MAX_BLOCK_SIZE;
    cout<<"Total number of Blocks required: "<<block_in_Grid<<endl;


    //Parallel Bitonic Sort start here...
    //Parallel code CUDA bitonic function call
    for (int k = 2; k <= nearest_pow2; k *= 2) 
    {
        // coming back to the host between values of k acts as a barrier
		// note that in later hardware (compute capabilty >= 7.0), there is a cuda::barrier avaliable
        for (int j = k>>1; j > 0; j = j>>1) 
        {
            bitonic<<<block_in_Grid, threads_per_Block>>>(d_data, j, k);
            hipDeviceSynchronize();
        }
	}

    //Copy the data from device to host memory
    hipMemcpy(data, d_data, array_Size_bytes, hipMemcpyDeviceToHost);

    //Validate Data is it sorted or not
    validate_Data(data, nearest_pow2);


    //Parallel sum or scan start here....
    // a array of flot values which contains parallel sum of a block of data
    float* d_sums;
    hipMalloc(&d_sums , block_in_Grid);

    //CUDA scan function scans and add data in sums which later on accumulate
    scan<<< block_in_Grid, threads_per_Block >>> (d_data, d_sums, nearest_pow2, block_in_Grid);
    hipDeviceSynchronize();

    // accumulate prefix sums of each block
    accum<<< block_in_Grid, threads_per_Block >>> (d_data, d_sums, nearest_pow2, block_in_Grid);
    hipDeviceSynchronize();

    //Copy the data from device to host memory
    hipMemcpy(data, d_data, array_Size_bytes, hipMemcpyDeviceToHost);


    //Write data in Result.csv file
    writeCSV(data, number_of_rows, output_filename);
    cout<<number_of_rows<<" Rows written in file: "<<output_filename<<endl;

    //Free used memory
    cout<<"* * *  Memory Free * * *"<<endl;
    hipFree(d_data);
    hipFree(d_sums);
    delete []data;
    return 0;
}

/*=========================================================FUNCTION DEFINATION===================================================================*/
//function to find next nearest power of 2
int nearestPowerOfTwo(int n) 
{
    // If n is already a power of 2, return n
    if (ceil(log2(n)) == floor(log2(n))) 
        return n;

    // Find the next power of 2 and the previous power of 2
    int nextPowerOfTwo = pow(2, ceil(log2(n)));

    // Return the nearest power of 2 
    return nextPowerOfTwo;
}

//function to skip entered number of lines
void skipLines(fstream& file, int numLines) 
{
    string line;
    for (int i = 0; i < numLines; i++) 
    {
        if (!std::getline(file, line)) 
            break;  // we've reached the end of the file before skipping all the lines
    }
}

//function to calculate number of rows
int calcrows(string filename)
{
    fstream fin;

    string line;

    fin.open(filename, ios::in);
    if (fin.fail())
    {
        cerr<<"Unable to open file"<<filename<<endl;
        exit(1);
    }

    //for loop to skip given number of lines
    int number_of_line_skip = 1;
    skipLines(fin, number_of_line_skip);

    int number_of_rows = 0;
    while (getline(fin, line)) 
    {
        number_of_rows++;
    }

    //Close file
    fin.close();
    return number_of_rows;
}

//Function to set data into structure
Data* ReadCSV(int& number_of_rows)
{
    // File pointer
    fstream fin;

    //variable for getting a line from file
    string line;
   
    // Open the file
    fin.open(filename, ios::in);
    if (fin.fail())
    {
        cerr<<"Unable to open file"<<filename<<endl;
        exit(1);
    }

    //convert number of rows to nearest power of 2 
    int nearest_pow2 = nearestPowerOfTwo(number_of_rows);

    //Object of Structure
    Data* data = new Data[nearest_pow2];

    int number_of_line_skip = 1;
    skipLines(fin, number_of_line_skip);

    for (int i = 0; i < number_of_rows; i++)
    {
        getline(fin, line);
        sscanf(line.c_str(), "%f,%f", &data[i].x, &data[i].y);
        data[i].row_num = i+1; 
        data[i].cumulative_y = 0.0;
    }
    
    for (int i = number_of_rows; i < nearest_pow2; i++)
    {
        data[i].x = FLT_MAX;
        data[i].y = 0;
        data[i].cumulative_y = 0.0;
        data[i].row_num = i+1;
    }
    fin.close();
    return data;
}

//Function to write all the data into csv file
void writeCSV(Data* data, int& number_of_rows, string filename) 
{
    ofstream outfile;
    outfile.open(filename);

    outfile << "Row Number,Value of X,Value of Y, Cumilative Y\n";

    for (int i = 0; i < number_of_rows; i++) 
    {
        outfile << data[i].row_num << ","
                << data[i].x << ","
                << data[i].y << ","
                << data[i].cumulative_y << "\n";
    }

    outfile.close();
}

//Function to validate the sorting of x elements
void validate_Data(Data* data, int& number_of_rows)
{
    for (int i = 0; i < number_of_rows - 1; i++)
    {
        if (data[i].x > data[i+1].x)
        {
            cout<<"Data Not Sorted Properly. Element: "<<i<<endl<<data[i].x<<" > "<<data[i+1].x<<endl;
            return ;
        }
    }
    cout<<"Data Sorted Properly....."<<endl;
}